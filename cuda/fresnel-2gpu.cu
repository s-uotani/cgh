#include <stdio.h>
#include <math.h>
#include <stdint.h>		//uint32_tは符号なしintで4バイトに指定
#include <stdlib.h> 	//記憶域管理を使うため
#include <hip/hip_runtime.h>
#include <omp.h>

//記号定数として横幅と縦幅を定義
#define width 1024
#define heigth 1024
#define pixel width*heigth


/*--------------------bmpの構造体--------------------*/
#pragma pack(push,1)
typedef struct tagBITMAPFILEHEADER{	//構造体BITMAPFILEHEADERはファイルの先頭に来るもので，サイズは14 byte
	unsigned short	bfType;			//bfTypeは，bmp形式であることを示すため，"BM"が入る
	uint32_t 		bfSize;			//bfsizeは，ファイル全体のバイト数
	unsigned short	bfReserved1;	//bfReserved1と2は予約領域で，0になる
	unsigned short	bfReserved2;
	uint32_t		bf0ffBits;		//bf0ffBitsは先頭から画素データまでのバイト数
}BITMAPFILEHEADER;

#pragma pack(pop)
typedef struct tagBITMAPINFOHEADER{		//BITMAPINFOHEADERはbmpファイルの画像の情報の構造体で，サイズは40 byte
	uint32_t		biSize;				//画像のサイズ
	uint32_t		biWidth;			//横の画素数
	uint32_t		biHeight;			//縦の画素数
	unsigned short	biPlanes;			//1
	unsigned short	biBitCount;			//一画素あたりの色の数のbit数．今回は8
	uint32_t		biCompression;		//圧縮タイプを表す．bmpは非圧縮なので0
	uint32_t		biSizeImage;		//bmp配列のサイズを表す．biCompression=0なら基本的に0
	uint32_t		biXPelsPerMeter;	//biXPelsPerMeterとbiYPelsPerMeterは基本的に0
	uint32_t		biYPelsPerMeter;
	uint32_t		biCirUsed;			//0
	uint32_t		biCirImportant;		//0
}BITMAPINFOHEADER;

typedef struct tagRGBQUAD{
	unsigned char	rgbBlue;
	unsigned char	rgbGreen;
	unsigned char	rgbRed;
	unsigned char	rgbReserved;
}RGBQUAD;
/*--------------------------------------------------*/


/*--------------------フレネル近似のカーネル関数--------------------*/
//上半分を計算すｓるカーネル
__global__ void fresnel_gpu_0(int *x_d, int *y_d, float *z_d, float *lumi_intensity_d){
    int i, j, k;
	int adr;
	float xx, yy;

    j = blockDim.x*blockIdx.x+threadIdx.x;	//widthのループの置き換え
	i = blockDim.y*blockIdx.y+threadIdx.y;	//heigthのループの置き換え
	adr = i*width+j;

	float wave_len = 0.633F;		//光波長
	float wave_num = M_PI/wave_len;	//波数の2分の1

	for (k=0; k<284; k++) {
		xx = ((float)j-x_d[k])*((float)j-x_d[k]);
		yy = ((float)i-y_d[k])*((float)i-y_d[k]);
		lumi_intensity_d[adr] = lumi_intensity_d[adr]+__cosf(wave_num*(xx+yy)*z_d[k]);
	}
}
//下半分を計算すｓるカーネル
__global__ void fresnel_gpu_1(int *x_d, int *y_d, float *z_d, float *lumi_intensity_d){
    int i, j, k;
	int adr;
	float xx, yy;

    j = blockDim.x*blockIdx.x+threadIdx.x;				//widthのループの置き換え
	i = blockDim.y*blockIdx.y+threadIdx.y+heigth*0.5;	//heigthのループの置き換え
	adr = (i-heigth*0.5)*width+j;

	float wave_len = 0.633F;		//光波長
	float wave_num = M_PI/wave_len;	//波数の2分の1

	for (k=0; k<284; k++) {
		xx = ((float)j-x_d[k])*((float)j-x_d[k]);
		yy = ((float)i-y_d[k])*((float)i-y_d[k]);
		lumi_intensity_d[adr] = lumi_intensity_d[adr]+__cosf(wave_num*(xx+yy)*z_d[k]);
	}
}
/*--------------------------------------------------*/


//画像生成用の配列
float lumi_intensity[pixel];	//光強度用の配列
float img_tmp0[pixel/2];
float img_tmp1[pixel/2];
unsigned char img[pixel];		//bmp用の配列


/*--------------------main関数--------------------*/
int main(){
	BITMAPFILEHEADER bmpFh;
	BITMAPINFOHEADER bmpIh;
	RGBQUAD rgbQ[256];

	//ホスト側の変数
	int i;
    int points;	//物体点
	float min = 0.0F, max = 0.0F, mid;	//2値化に用いる
	FILE *fp;

	//3Dファイルの読み込み
	fp = fopen("cube284.3d","rb");	//バイナリで読み込み
	if (!fp) {
		printf("3D file not found!\n");
		exit(1);
	}
	fread(&points, sizeof(int), 1, fp);	//データのアドレス，サイズ，個数，ファイルポインタを指定
	printf("the number of points is %d\n", points);

    //取り出した物体点を入れる配列
	int x[points];				//~~データを読み込むことで初めてこの配列が定義できる~~
	int y[points];
	float z[points];
	int x_buf, y_buf, z_buf;	//データを一時的に溜めておくための変数

	//各バッファに物体点座標を取り込み，ホログラム面と物体点の位置を考慮したデータを各配列に入れる
	for (i=0; i<points; i++) {
		fread(&x_buf, sizeof(int), 1, fp);
		fread(&y_buf, sizeof(int), 1, fp);
		fread(&z_buf, sizeof(int), 1, fp);

		x[i] = x_buf*40+width*0.5;	//物体点を離すために物体点座標に40を掛け，中心の座標を足す
		y[i] = y_buf*40+heigth*0.5;
		z[i] = 1.0F/(((float)z_buf)*40+10000.0F);
	}
	fclose(fp);


/*--------------------GPUによるCGH計算--------------------*/
	dim3 block(32,32/2,1);	//ブロックサイズ(スレッド数)の配置
    dim3 grid(ceil(width/block.x),ceil(heigth/(block.y*2)),1);	//グリッドサイズ(ブロック数)の配置
//	dim3 grid((width+block.x-1)/block.x,(heigth+block.y-1)/block.y,1);

	//デバイス側の変数
	int *x_d;
	int *y_d;
	float *z_d;
	float *lumi_intensity_d;

	omp_set_num_threads(2);
	#pragma omp parallel sections
	{	//"{"は次の行に書かないと大量にエラーが吐き出される！！
		#pragma omp section	//GPU0
		{
			//使用するデバイスを指定
			hipSetDevice(0);
			//デバイス側のメモリ確保
			hipMalloc((void**)&x_d, points*sizeof(int));
			hipMalloc((void**)&y_d, points*sizeof(int));
			hipMalloc((void**)&z_d, points*sizeof(float));
			hipMalloc((void**)&lumi_intensity_d, pixel/2*sizeof(float));
			//ホスト側からデバイス側にデータ転送
			hipMemcpy(x_d, x, points*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(y_d, y, points*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(z_d, z, points*sizeof(float), hipMemcpyHostToDevice);
			//カーネル関数の起動
			fresnel_gpu_0<<< grid, block >>>(x_d, y_d, z_d, lumi_intensity_d);
			hipDeviceSynchronize();	//同期
			//デバイス側からホスト側にデータ転送
			hipMemcpy(img_tmp0, lumi_intensity_d, pixel/2*sizeof(float), hipMemcpyDeviceToHost);
			//デバイスのメモリ解放
			hipFree(x_d);
			hipFree(y_d);
			hipFree(z_d);
			hipFree(lumi_intensity_d);
		}
		#pragma omp section	//GPU2
		{
			//使用するデバイスを指定
			hipSetDevice(2);
			//デバイス側のメモリ確保
			hipMalloc((void**)&x_d, points*sizeof(int));
			hipMalloc((void**)&y_d, points*sizeof(int));
			hipMalloc((void**)&z_d, points*sizeof(float));
			hipMalloc((void**)&lumi_intensity_d, pixel/2*sizeof(float));
			//ホスト側からデバイス側にデータ転送
			hipMemcpy(x_d, x, points*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(y_d, y, points*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(z_d, z, points*sizeof(float), hipMemcpyHostToDevice);
			//カーネル関数の起動
			fresnel_gpu_1<<< grid, block >>>(x_d, y_d, z_d, lumi_intensity_d);
			hipDeviceSynchronize();
			//デバイス側からホスト側にデータ転送
			hipMemcpy(img_tmp1, lumi_intensity_d, pixel/2*sizeof(float), hipMemcpyDeviceToHost);
			//デバイスのメモリ解放
			hipFree(x_d);
			hipFree(y_d);
			hipFree(z_d);
			hipFree(lumi_intensity_d);
		}
	}
/*--------------------------------------------------*/

	//1つの配列に統合
	for (i=0; i<pixel/2; i++) {
		lumi_intensity[i] = img_tmp0[i];
		lumi_intensity[i+pixel/2] = img_tmp1[i];
	}

	//最大値，最小値を求める
	for (i=0; i<pixel; i++) {
		if (min>lumi_intensity[i]) {
			min = lumi_intensity[i];
		}
		if (max<lumi_intensity[i]) {
			max = lumi_intensity[i];
		}
	}
	mid = (min+max)/2;	//中間値（閾値）を求める

	//各々の光強度配列の値を中間値と比較し，2値化する
	for (i=0; i<pixel; i++) {
		if (lumi_intensity[i]<mid) {
			img[i] = 0;
		}
		else{
			img[i] = 255;
		}
	}


/*--------------------BMP関連--------------------*/
		//BITMAPFILEHEADERの構造体
		bmpFh.bfType		= 19778;	//'B'=0x42,'M'=0x4d,'BM'=0x4d42=19778
		bmpFh.bfSize		= 14+40+1024+(pixel);	//1024はカラーパレットのサイズ．256階調で4 byte一組
		bmpFh.bfReserved1	= 0;
		bmpFh.bfReserved2	= 0;
		bmpFh.bf0ffBits		= 14+40+1024;
		//BITMAPINFOHEADERの構造体
		bmpIh.biSize			= 40;
		bmpIh.biWidth			= width;
		bmpIh.biHeight			= heigth;
		bmpIh.biPlanes			= 1;
		bmpIh.biBitCount		= 8;
		bmpIh.biCompression		= 0;
		bmpIh.biSizeImage		= 0;
		bmpIh.biXPelsPerMeter	= 0;
		bmpIh.biYPelsPerMeter	= 0;
		bmpIh.biCirUsed			= 0;
		bmpIh.biCirImportant	= 0;
		//RGBQUADの構造体
		for (i=0; i<256; i++) {
			rgbQ[i].rgbBlue		= i;
			rgbQ[i].rgbGreen	= i;
			rgbQ[i].rgbRed		= i;
			rgbQ[i].rgbReserved	= 0;
		}
/*--------------------------------------------------*/


	fp = fopen("fresnel-2gpu.bmp","wb");	//宣言したfpと使用するファイル名，その読み書きモードを設定．バイナリ(b)で書き込み(w)
	fwrite(&bmpFh, sizeof(bmpFh), 1, fp);	//書き込むデータのアドレス，データのサイズ，データの個数，ファイルのポインタを指定
	fwrite(&bmpIh, sizeof(bmpIh), 1, fp);	//(&bmpFh.bfType, sizeof(bmpFh.bfType), 1, fp);というように個別に書くことも可能
	fwrite(&rgbQ[0], sizeof(rgbQ[0]), 256, fp);
	fwrite(img, sizeof(unsigned char), pixel, fp);	//bmpに書き込み
	printf("'fresnel-2gpu.bmp' was saved.\n\n");
	fclose(fp);

	return 0;
}
/*--------------------main関数--------------------*/
